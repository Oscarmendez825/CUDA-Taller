#include <iostream>
#include <hip/hip_runtime.h>

const int N = 1000; // Tamaño de los arreglos

// Kernel para el producto punto de dos arreglos
// a y b son los arreglos de entrada
// result es el arreglo de salida
// n es el tamaño de los arreglos
__global__ void dotProductKernel(const float* a, const float* b, float* result, int n) {
    
    // Calcula el índice global
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Variable compartida para almacenar la suma parcial en el bloque
    __shared__ float partialSum[256];

    // Inicializa la suma parcial en el bloque
    partialSum[threadIdx.x] = 0.0f;

    // Calcula la suma parcial en el bloque
    while (tid < n) {
        partialSum[threadIdx.x] += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Asegura que se hayan completado la sumas parciales antes de continuar.
    __syncthreads();

    // Se reduce el número de elementos a la mitad en cada iteración
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + stride];
        }
        // Asegura que se hayan completado las sumas parciales antes de continuar.
        __syncthreads();
    }

    // El primer thread de cada bloque almacena el resultado final 
    // en el vector de resultados
    if (threadIdx.x == 0) {
        atomicAdd(result, partialSum[0]);
    }
}

int main() {

    // Tamaño de los arreglos en bytes
    const int arraySize = N * sizeof(float);
    // Arreglos en el host
    float *h_a, *h_b, *h_result;
    // Arreglos en el dispositivo
    float *d_a, *d_b, *d_result;

    // Asigna memoria en el host
    h_a = (float*)malloc(arraySize);
    h_b = (float*)malloc(arraySize);
    h_result = (float*)malloc(sizeof(float));

    // Inicializa arreglos en el host
    // Todos los elementos de los arreglos son 2.0f y 4.0f respectivamente
    for (int i = 0; i < N; ++i) {
        h_a[i] = 2.0f;
        h_b[i] = 4.0f;
    }

    // Asigna memoria en el dispositivo 
    hipMalloc((void**)&d_a, arraySize);
    hipMalloc((void**)&d_b, arraySize);
    hipMalloc((void**)&d_result, sizeof(float));

    // Copia datos del host al dispositivo
    hipMemcpy(d_a, h_a, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arraySize, hipMemcpyHostToDevice);
    // Inicializa el resultado en el dispositivo
    hipMemset(d_result, 0, sizeof(float));

    // Configura dimensiones del bloque 
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Lanza el kernel
    dotProductKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_result, N);

    // Copia el resultado de vuelta al host
    // El resultado se encuentra en el dispositivo
    // por lo que se debe copiar de vuelta al host
    hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Imprime el resultado
    std::cout << "Resultado del producto punto: " << *h_result << std::endl;

    // Libera memoria en el host y el dispositivo
    free(h_a);
    free(h_b);
    free(h_result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
